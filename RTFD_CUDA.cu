#include "hip/hip_runtime.h"
#include "project.h"

#define DEVICE_CHECK_OOB if(i < d_pad || i >= d_dim-d_pad || j < d_pad || j >= d_dim-d_pad) {printf("DEVICE BOUNDS ERROR: Cell (%i,%i) OOB at line %d\n", i,j, __LINE__);}
#define INBOUNDS(i,j) (i>=d_pad && i<d_dim-d_pad && j>=d_pad && j<d_dim-d_pad)
#define FOR_JOBS_IN_BOUNDS uint32_t i = (blockIdx.x*blockDim.x*d_CELLSPERTHREAD)+(threadIdx.x*d_CELLSPERTHREAD); uint32_t const j = blockIdx.y*blockDim.y+threadIdx.y; for (uint32_t jobnum = 0; jobnum < d_CELLSPERTHREAD; jobnum++) { if INBOUNDS(i,j) 
#define END_JOBS i++; }	
#define DIX(i,j) ((i)+(d_dim)*(j))


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace CUDA { // CUDA implementation

	//Device globals
	__device__ uint32_t d_dim, d_pad, d_bnd, d_N, d_BSIZE, d_CELLSPERTHREAD;

	//Host globals
	dim3 blockSize;
	uint32_t bx;
	uint32_t by;
	dim3 gridSize;
	dim3 threadCount;

	uint32_t BSIZE = 32;
	uint32_t CELLSPERTHREAD = 4;

	//Device array copies
	float *d_x, *d_x0, *d_u, *d_v, *d_u0, *d_v0; 	
	size_t size;

	//
	// CUDA Initialization/Teardown
	//

	__host__
	int allocate_data_cuda_pinned(void** ptr, size_t size)
	{
		if (hipHostAlloc(ptr, size, hipHostMallocDefault) == hipSuccess) {
			return 1;
		}
		else return 0;
	}

	__global__
	void gpu_init_cuda_globals(uint32_t N, uint32_t dim, uint32_t bnd, uint32_t pad, uint32_t BSIZE, uint32_t CELLSPERTHREAD)
	{
		d_N = N;
		d_dim = dim;
		d_bnd = bnd;
		d_pad = pad;
		d_BSIZE = BSIZE;
		d_CELLSPERTHREAD = CELLSPERTHREAD;
	}

	__host__
	void init_cuda_globals(uint32_t N, uint32_t dim, uint32_t bnd, uint32_t pad)
	{
		gpu_init_cuda_globals<<<1,1>>>(N, dim, bnd, pad, BSIZE, CELLSPERTHREAD);

		blockSize = dim3(BSIZE, BSIZE);
		bx = (int)ceil((dim) / blockSize.x);
		by = (int)ceil((dim) / blockSize.y);
		gridSize = dim3(bx, by);
		threadCount = dim3(BSIZE/CELLSPERTHREAD, BSIZE);

		size = sizeof(float)*array_size;
		gpuErrchk(hipMalloc((void**)&d_x, size));
		gpuErrchk(hipMalloc((void**)&d_x0, size));
		gpuErrchk(hipMalloc((void**)&d_u, size	));
		gpuErrchk(hipMalloc((void**)&d_v, size	));
		gpuErrchk(hipMalloc((void**)&d_u0, size));
		gpuErrchk(hipMalloc((void**)&d_v0, size));
	}

	__host__
	void dealloc_cuda_globals()
	{
		gpuErrchk(hipFree(d_x));
		gpuErrchk(hipFree(d_x0));
		gpuErrchk(hipFree(d_u));
		gpuErrchk(hipFree(d_v));
		gpuErrchk(hipFree(d_u0));
		gpuErrchk(hipFree(d_v0));
	}

	//
	// Device Kernels
	//

	__global__
	void gpu_project_1(float mul, float* u, float* v, float* p, float* div) {
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			const float sub1 = u[DIX(i + 1, j)] - u[DIX(i - 1, j)];
			const float sub2 = v[DIX(i, j + 1)] - v[DIX(i, j - 1)];
			const float sum = sub1 + sub2;
			const float result = sum * mul;
			div[DIX(i, j)] = result;
			p[DIX(i, j)] = 0;
		} END_JOBS
	}

	__global__
	void gpu_project_2(float* p, float* u, float* v) {
		const uint32_t mul = 0.5f * d_N;
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			const float a = p[DIX(i + 1, j)] - p[DIX(i - 1, j)];
			u[DIX(i, j)] = u[DIX(i, j)] - mul*a;
			const float b = p[DIX(i, j + 1)] - p[DIX(i, j - 1)];
			v[DIX(i, j)] = v[DIX(i, j)] - mul*b;
		} END_JOBS
	}

	__global__
	void gpu_lin_solve(float* x, float* x0, const float a, const float c) {
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			const float left = x[DIX(i - 1, j)];
			const float right = x[DIX(i + 1, j)];
			const float above = x[DIX(i, j + 1)];
			const float below = x[DIX(i, j - 1)];
			const float sum = left + right + above + below;
			const float prod = sum * a;
			const float addOrig = prod + x0[DIX(i, j)];
			const float result = addOrig / c;
			x[DIX(i, j)] = result;
		} END_JOBS
	}

	__global__
	void gpu_advect(float *d_d, float *d_d0, float *d_u, float *d_v, const float dt){
		uint32_t i0, j0, i1, j1;
		float x, y, s0, t0, s1, t1, dt0;
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			dt0 = dt * d_N;
			x = i - dt0 * d_u[DIX(i, j)];
			y = j - dt0 * d_v[DIX(i, j)];
			if (x < 0.5f)
				x = 0.5f;
			if (x > d_N + 0.5f)
				x = d_N + 0.5f;
			i0 = (int)x;
			i1 = i0 + 1;
			if (y < 0.5f)
				y = 0.5f;
			if (y > d_N + 0.5f)
				y = d_N + 0.5f;
			j0 = (int)y;
			j1 = j0 + 1;
			s1 = x - i0;
			s0 = 1 - s1;
			t1 = y - j0;
			t0 = 1 - t1;
			d_d[DIX(i, j)] = s0 * (t0 * d_d0[DIX(i0, j0)] + t1 * d_d0[DIX(i0, j1)]) +
				s1 * (t0 * d_d0[DIX(i1, j0)] + t1 * d_d0[DIX(i1, j1)]);
		} END_JOBS
	}

	__global__
	void gpu_advect_velstep_u(float *d_u, float *d_u0, float *d_v0, const float dt){
		uint32_t i0, j0, i1, j1;
		float x, y, s0, t0, s1, t1, dt0;
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			dt0 = dt * d_N;
			x = i - dt0 * d_u0[DIX(i, j)];
			y = j - dt0 * d_v0[DIX(i, j)];
			if (x < 0.5f)
				x = 0.5f;
			if (x > d_N + 0.5f)
				x = d_N + 0.5f;
			i0 = (int)x;
			i1 = i0 + 1;
			if (y < 0.5f)
				y = 0.5f;
			if (y > d_N + 0.5f)
				y = d_N + 0.5f;
			j0 = (int)y;
			j1 = j0 + 1;
			s1 = x - i0;
			s0 = 1 - s1;
			t1 = y - j0;
			t0 = 1 - t1;
			d_u[DIX(i, j)] = s0 * (t0 * d_u0[DIX(i0, j0)] + t1 * d_u0[DIX(i0, j1)]) +
				s1 * (t0 * d_u0[DIX(i1, j0)] + t1 * d_u0[DIX(i1, j1)]);
		} END_JOBS
	}
	__global__
	void gpu_advect_velstep_v(float *d_v, float *d_u0, float *d_v0, const float dt){
		uint32_t i0, j0, i1, j1;
		float x, y, s0, t0, s1, t1, dt0;
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			dt0 = dt * d_N;
			x = i - dt0 * d_u0[DIX(i, j)];
			y = j - dt0 * d_v0[DIX(i, j)];
			if (x < 0.5f)
				x = 0.5f;
			if (x > d_N + 0.5f)
				x = d_N + 0.5f;
			i0 = (int)x;
			i1 = i0 + 1;
			if (y < 0.5f)
				y = 0.5f;
			if (y > d_N + 0.5f)
				y = d_N + 0.5f;
			j0 = (int)y;
			j1 = j0 + 1;
			s1 = x - i0;
			s0 = 1 - s1;
			t1 = y - j0;
			t0 = 1 - t1;
			d_v[DIX(i, j)] = s0 * (t0 * d_v0[DIX(i0, j0)] + t1 * d_v0[DIX(i0, j1)]) +
				s1 * (t0 * d_v0[DIX(i1, j0)] + t1 * d_v0[DIX(i1, j1)]);
		} END_JOBS
	}

	__global__
	void gpu_set_bnd(uint32_t b, float *x)
	{
		uint32_t const n = blockIdx.x*blockDim.x+threadIdx.x;
		const uint32_t ub = d_N+d_pad;
		const uint32_t lb = d_pad-1;
		if(n < ub && n > lb){
			x[DIX(lb, n)] = (b == 1) ? -x[DIX(lb+1, n)] : x[DIX(lb+1, n)];	
			x[DIX(ub, n)] = (b == 1) ? -x[DIX(ub-1, n)] : x[DIX(ub-1, n)];
			x[DIX(n, lb)] = (b == 2) ? -x[DIX(n, lb+1)] : x[DIX(n, lb+1)];
			x[DIX(n, ub)] = (b == 2) ? -x[DIX(n, ub-1)] : x[DIX(n, ub-1)];

		}

		//Omitting corner cells for simplicity's sake, since I haven't noticed any anomalies without them.
		//x[DIX(d_pad-1, d_pad-1)] = 0.5f * (x[DIX(d_pad, d_pad-1)] + x[DIX(d_pad-1, d_pad)]);
		//x[DIX(d_pad-1, d_N + d_pad)] = 0.5f * (x[DIX(d_pad, d_N+d_pad)] + x[DIX(d_pad-1, d_N+d_pad-1)]);
		//x[DIX(N+d_pad, d_pad-1)] = 0.5f * (x[DIX(N+d_pad-1,d_pad-1)] + x[DIX(N+d_pad, d_pad)]);
		//x[DIX(N+d_pad, d_N+d_pad)] = 0.5f * (x[DIX(N+d_pad-1, d_N+d_pad)] + x[DIX(N+d_pad, d_N+d_pad-1)]);
	}

	__global__
	void gpu_add_source(float *x, float *s, const float dt){
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			x[DIX(i,j)] += dt * s[DIX(i,j)];
		} END_JOBS
	}

	void lin_solve(uint32_t b, float *d_x, float *d_x0, float a, float c){
		uint32_t k;
		for (k = 0; k < 20; k++)
		{
			CUDA::gpu_lin_solve<<<gridSize, threadCount>>>(d_x, d_x0, a, c);
			CUDA::gpu_set_bnd<<<bx, BSIZE>>>(b, d_x);
		}
	}

	void diffuse(uint32_t b, float *d_x, float *d_x0, float diff, float dt)
	{
		float a = dt * diff * N * N;
		CUDA::lin_solve(b, d_x, d_x0, a, 1 + 4 * a);
	}

	void project(float *d_u, float *d_v, float *d_p, float *d_div)
	{
		const float mul = -0.5f/(float)N;
		CUDA::gpu_project_1<<<gridSize, threadCount>>>(mul, d_u, d_v, d_p, d_div);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_div);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_p);
		uint32_t k;
		for (k = 0; k < 20; k++)
		{
			CUDA::gpu_lin_solve<<<gridSize, threadCount>>>(d_p, d_div, 1, 4);
			CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_p);
		}
		CUDA::gpu_project_2<<<gridSize, threadCount>>>(d_p, d_u, d_v);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(1, d_u);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(2, d_v);
	}

	void combined_step(float *u, float *v, float *u0, float *v0, float visc, float *x, float *x0, float diff, float dt) {
		
		/*
		Prepare device memory
		*/
		gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_u0, u0, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_v0, v0, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_x, x, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_x0, x0, size, hipMemcpyHostToDevice	));

		/*
		Compute velocity step
		*/
		CUDA::gpu_add_source<<<gridSize, threadCount>>>(d_u, d_u0, dt);
		SWAP(d_u0, d_u);
		CUDA::diffuse(1, d_u, d_u0, visc, dt);
		CUDA::gpu_add_source<<<gridSize, threadCount>>>(d_v, d_v0, dt);
		SWAP(d_v0, d_v);
		CUDA::diffuse(2, d_v, d_v0, visc, dt);
		CUDA::project(d_u, d_v, d_u0, d_v0);
		SWAP(d_u0, d_u);
		SWAP(d_v0, d_v);
		CUDA::gpu_advect_velstep_u<<<gridSize, threadCount>>>(d_u, d_u0, d_v0, dt);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(1, d_u);
		CUDA::gpu_advect_velstep_v<<<gridSize, threadCount>>>(d_v, d_u0, d_v0, dt);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(2, d_v);
		CUDA::project(d_u, d_v, d_u0, d_v0);

		/*
		Compute density step
		*/
		CUDA::gpu_add_source<<<gridSize, threadCount>>>(d_x, d_x0, dt);
		SWAP(d_x0, d_x);
		CUDA::diffuse(0, d_x, d_x0, diff, dt);
		SWAP(d_x0, d_x);
		CUDA::gpu_advect<<<gridSize, threadCount>>>(d_x, d_x0, d_u, d_v, dt);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_u);

		/*
		Return result to CPU memory
		*/
		gpuErrchk(hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(v, d_v, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(x0, d_x0, size, hipMemcpyDeviceToHost));

	}


	//
	// DEBUG Functions
	//

	__global__
		void gpu_full_address_test(float* in, float* out) {
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			float blue = ((blockIdx.x+blockIdx.y) % 2) * 1.0f;
		float red = ((threadIdx.y % d_BSIZE)/(float)d_BSIZE) * 1.0f;
		in[DIX(i, j)] = blue;
		out[DIX(i, j)] = red;
		} END_JOBS
	}

	__host__
		void full_address_test(float* in, float* out){
		float *d_in, *d_out;
		gpuErrchk(hipMalloc((void**)&d_in, size));
		gpuErrchk(hipMalloc((void**)&d_out, size));
		gpuErrchk(hipMemcpy(d_in, in, size, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_out, out, size, hipMemcpyHostToDevice));

		/*
		Compute solution
		*/

		CUDA::gpu_full_address_test<<<gridSize, threadCount>>>(d_in, d_out);

		/*
		Return result to CPU memory
		*/

		gpuErrchk(hipMemcpy(in, d_in, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(d_in));
		gpuErrchk(hipFree(d_out));
	}

}

