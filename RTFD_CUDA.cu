#include "hip/hip_runtime.h"
#include "project.h"

namespace CUDA { // CUDA implementation
	
	__global__
	void gpu_lin_solve(float* x, float* x0, float a, float c, int N, int pad, int bnd) {
		int const dim = N + bnd;
		int const i = blockIdx.x*blockDim.x+threadIdx.x;
		int const j = blockIdx.y*blockDim.y+threadIdx.y;
		if (i < dim-pad && j < dim-pad && i > pad && j > pad)
		{
			const float left = x[IX(i - 1, j)];
			const float right = x[IX(i + 1, j)];
			const float above = x[IX(i, j + 1)];
			const float below = x[IX(i, j - 1)];
			const float sum = left + right + above + below;
			const float prod = sum * a;
			const float addOrig = prod + x0[IX(i, j)];
			const float result = addOrig / c;
			x[IX(i, j)] = result;
		}
	}

	__global__
	void gpu_set_bnd(int N, int pad, int bnd, uint32_t b, float *x)
	{
		//int const dim = N + bnd;
		int const i = blockIdx.x*blockDim.x+threadIdx.x;
		int const j = blockIdx.y*blockDim.y+threadIdx.y;

		if (i == pad-1) {
			x[IX(i, j)] = (b == 1) ? -x[IX(i+1, j)] : x[IX(i+1, j)];
		}		
		
		else if (i == N+pad) {
			x[IX(i, j)] = (b == 1) ? -x[IX(i-1, j)] : x[IX(i-1, j)];
		}

		else if (j == pad-1) {
			x[IX(i, j)] = (b == 2) ? -x[IX(i, j+1)] : x[IX(i, j+1)];
		}		

		else if (j == N+pad) {
			x[IX(i, j)] = (b == 2) ? -x[IX(i, j-1)] : x[IX(i, j-1)];
		}

		//x[IX(pad-1, pad-1)] = 0.5f * (x[IX(pad, pad-1)] + x[IX(pad-1, pad)]);
		//x[IX(pad-1, N + pad)] = 0.5f * (x[IX(pad, N+pad)] + x[IX(pad-1, N+pad-1)]);
		//x[IX(N+pad, pad-1)] = 0.5f * (x[IX(N+pad-1,pad-1)] + x[IX(N+pad, pad)]);
		//x[IX(N+pad, N+pad)] = 0.5f * (x[IX(N+pad-1, N+pad)] + x[IX(N+pad, N+pad-1)]);
	}

	__host__
	void add_source(uint32_t N, float *x, float *s, float dt)
	{
		uint32_t i, size = (N + bnd) * (N + bnd);
		for (i = 0; i < size; i++)
			x[i] += dt * s[i];
	}


	void lin_solve(uint32_t N, uint32_t b, float *x, float *x0, float a, float c)
	{
		//const int blocksize = 64;
		auto const size = sizeof(float)*array_size;
		//printf("Size = %i\n", (int)size);
		//printf("array_size = %i\n", (int)array_size);

		//auto const num_blocks = ceil(array_size / static_cast<int>(blocksize));

		float *dev_x, *dev_x0;
		hipMalloc((void**)&dev_x, size);
		hipMalloc((void**)&dev_x0, size);
		
		dim3 blockSize = dim3(8, 8);

		int dim = N + bnd;
		int bx = (dim + blockSize.x - 1) / blockSize.x;
		int by = (dim + blockSize.y - 1) / blockSize.y;
		dim3 gridSize = dim3(bx, by);

		uint32_t k;

		hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice);
		hipMemcpy(dev_x0, x0, size, hipMemcpyHostToDevice);

		for (k = 0; k < 20; k++)
		{
			CUDA::gpu_lin_solve<<<gridSize, blockSize>>>(dev_x, dev_x0, a, c, N, pad, bnd);
			CUDA::gpu_set_bnd<<<gridSize, blockSize>>>(N, pad, bnd, b, dev_x);
		}

		hipMemcpy(x, dev_x, size, hipMemcpyDeviceToHost);
		hipMemcpy(x0, dev_x0, size, hipMemcpyDeviceToHost);

		hipFree(dev_x);
		hipFree(dev_x0);
	}


	void set_bnd(uint32_t N, uint32_t b, float *x)
	{
		uint32_t i;
		uint32_t pad = bnd/2;
		for (i = pad; i < N+pad; i++)
		{
			x[IX(pad-1, i)] = b == 1 ? -x[IX(pad, i)] : x[IX(pad, i)];
			x[IX(N+pad, i)] = b == 1 ? -x[IX(N+pad-1, i)] : x[IX(N+pad-1, i)];
			x[IX(i, pad-1)] = b == 2 ? -x[IX(i, pad)] : x[IX(i, pad)];
			x[IX(i, N+pad)] = b == 2 ? -x[IX(i, N+pad-1)] : x[IX(i, N+pad-1)];
		}
		x[IX(pad-1, pad-1)] = 0.5f * (x[IX(pad, pad-1)] + x[IX(pad-1, pad)]);
		x[IX(pad-1, N + pad)] = 0.5f * (x[IX(pad, N+pad)] + x[IX(pad-1, N+pad-1)]);
		x[IX(N+pad, pad-1)] = 0.5f * (x[IX(N+pad-1,pad-1)] + x[IX(N+pad, pad)]);
		x[IX(N+pad, N+pad)] = 0.5f * (x[IX(N+pad-1, N+pad)] + x[IX(N+pad, N+pad-1)]);
	}


	//void lin_solve(uint32_t N, uint32_t b, float* x, float* x0, float a, float c)
	//{
	//	uint32_t i, j, k;
	//	for (k = 0; k < 20; k++){
	//		for ( j=4; j<=N+4; j++ ) { 
	//			for ( i=4; i<=N+4; i+=4 ) {
	//				x[IX(i, j)] = (x0[IX(i, j)] + a * (x[IX(i - 1, j)] + x[IX(i + 1, j)] + x[IX(i, j - 1)] + x[IX(i, j + 1)])) / c;
	//			}
	//		}
	//		CUDA::set_bnd(N, b, x);
	//	}
	//}

	void diffuse(uint32_t N, uint32_t b, float *x, float *x0, float diff, float dt)
	{
		float a = dt * diff * N * N;
		CUDA::lin_solve(N, b, x, x0, a, 1 + 4 * a);
	}

	void advect(uint32_t N, uint32_t b, float *d, float *d0, float *u, float *v, float dt)
	{
		uint32_t i, j, i0, j0, i1, j1;
		float x, y, s0, t0, s1, t1, dt0;

		dt0 = dt * N;
		FOR_EACH_CELL
			x = i - dt0 * u[IX(i, j)];
			y = j - dt0 * v[IX(i, j)];
			if (x < 0.5f)
				x = 0.5f;
			if (x > N + 0.5f)
				x = N + 0.5f;
			i0 = (int)x;
			i1 = i0 + 1;
			if (y < 0.5f)
				y = 0.5f;
			if (y > N + 0.5f)
				y = N + 0.5f;
			j0 = (int)y;
			j1 = j0 + 1;
			s1 = x - i0;
			s0 = 1 - s1;
			t1 = y - j0;
			t0 = 1 - t1;
			d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) +
						s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
		END_FOR
		CUDA::set_bnd(N, b, d);
	}

	void project(uint32_t N, float *u, float *v, float *p, float *div)
	{
		uint32_t i, j;
		const float mul = -0.5f/(float)N;
		FOR_EACH_CELL
			const float sub1 = u[IX(i + 1, j)] - u[IX(i - 1, j)];
			const float sub2 = v[IX(i, j + 1)] - v[IX(i, j - 1)];
			const float sum = sub1 + sub2;
			const float result = sum * mul;
		 	div[IX(i, j)] = result;
			p[IX(i, j)] = 0;
		END_FOR
		CUDA::set_bnd(N, 0, div);
		CUDA::set_bnd(N, 0, p);

		CUDA::lin_solve(N, 0, p, div, 1, 4);

		FOR_EACH_CELL
			u[IX(i, j)] -= 0.5f * N * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
			v[IX(i, j)] -= 0.5f * N * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
		END_FOR
		CUDA::set_bnd(N, 1, u);
		CUDA::set_bnd(N, 2, v);
	}

	void dens_step(uint32_t N, float *x, float *x0, float *u, float *v, float diff, float dt)
	{
		CUDA::add_source(N, x, x0, dt);
		SWAP(x0, x);
		CUDA::diffuse(N, 0, x, x0, diff, dt);
		SWAP(x0, x);
		CUDA::advect(N, 0, x, x0, u, v, dt);
	}
	
	void vel_step(uint32_t N, float *u, float *v, float *u0, float *v0, float visc, float dt)
	{
		CUDA::add_source(N, u, u0, dt);
		SWAP(u0, u);
		CUDA::diffuse(N, 1, u, u0, visc, dt);
		CUDA::add_source(N, v, v0, dt);				
		SWAP(v0, v);
		CUDA::diffuse(N, 2, v, v0, visc, dt);
		CUDA::project(N, u, v, u0, v0);
		SWAP(u0, u);
		SWAP(v0, v);
		CUDA::advect(N, 1, u, u0, u0, v0, dt);
		CUDA::advect(N, 2, v, v0, u0, v0, dt);
		CUDA::project(N, u, v, u0, v0);
	}
}