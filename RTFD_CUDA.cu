#include "hip/hip_runtime.h"
#include "project.h"

#define BSIZE 16
#define JOBSPERTHREAD 4
#define DEVICE_CHECK_OOB if(i < d_pad || i >= d_dim-d_pad || j < d_pad || j >= d_dim-d_pad) {printf("DEVICE BOUNDS ERROR: Cell (%i,%i) OOB at line %d\n", i,j, __LINE__);}
#define INBOUNDS(i,j) (i>=d_pad && i<d_dim-d_pad && j>=d_pad && j<d_dim-d_pad)
#define FOR_JOBS_IN_BOUNDS for (uint32_t jobnum = 0; jobnum < JOBSPERTHREAD; jobnum++) { if INBOUNDS(i,j) 
#define END_JOBS i++; }	
#define DIX(i,j) ((i+d_pad)+(d_N+d_bnd)*(j))


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

namespace CUDA { // CUDA implementation

	//device globals
	__device__ uint32_t d_dim, d_pad, d_bnd, d_N;

	//host globals
	dim3 blockSize;
	uint32_t bx;
	uint32_t by;
	dim3 gridSize;
	dim3 threadCount;


	//
	// CUDA Initialization
	//
	__host__
	int allocate_data_cuda_pinned(void** ptr, size_t size)
	{
		if (hipHostAlloc(ptr, size, hipHostMallocDefault) == hipSuccess) {
			return 1;
		}
		else return 0;
	}

	__global__
	void gpu_init_cuda_globals(uint32_t N, uint32_t dim, uint32_t bnd, uint32_t pad)
	{
		d_dim = dim;
		d_pad = pad;
		d_bnd = bnd;
		d_N = N;
	}

	__host__
	void init_cuda_globals(uint32_t N, uint32_t dim, uint32_t bnd, uint32_t pad)
	{
		gpu_init_cuda_globals<<<1,1>>>(N, dim, bnd, pad);

		blockSize = dim3(BSIZE, BSIZE);
		bx = (int)ceil((dim) / blockSize.x);
		by = (int)ceil((dim) / blockSize.y);
		gridSize = dim3(bx, by);
		threadCount = dim3(BSIZE/JOBSPERTHREAD, BSIZE);
	}

	//
	// Device Kernels
	//

	__global__
	void gpu_project_1(float mul, float* u, float* v, float* p, float* div) {
		uint32_t i = blockIdx.x*(JOBSPERTHREAD*blockDim.x)+threadIdx.x*JOBSPERTHREAD;
		uint32_t const j = blockIdx.y*blockDim.y+threadIdx.y;

		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			const float sub1 = u[DIX(i + 1, j)] - u[DIX(i - 1, j)];
			const float sub2 = v[DIX(i, j + 1)] - v[DIX(i, j - 1)];
			const float sum = sub1 + sub2;
			const float result = sum * mul;
			div[DIX(i, j)] = result;
			p[DIX(i, j)] = 0;
		} END_JOBS
	}

	__global__
	void gpu_project_2(float* p, float* u, float* v) {
		uint32_t i = blockIdx.x*(JOBSPERTHREAD*blockDim.x)+threadIdx.x*JOBSPERTHREAD;
		const uint32_t j = blockIdx.y*blockDim.y+threadIdx.y;
		const uint32_t mul = 0.5f * d_N;
		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			const float a = p[DIX(i + 1, j)] - p[DIX(i - 1, j)];
			u[DIX(i, j)] = u[DIX(i, j)] - mul*a;
			const float b = p[DIX(i, j + 1)] - p[DIX(i, j - 1)];
			v[DIX(i, j)] = v[DIX(i, j)] - mul*b;
		} END_JOBS
	}

	__global__
	void gpu_lin_solve(float* x, float* x0, const float a, const float c) {
		uint32_t i = blockIdx.x*(JOBSPERTHREAD*blockDim.x)+threadIdx.x*JOBSPERTHREAD;
		uint32_t const j = blockIdx.y*blockDim.y+threadIdx.y;

		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			const float left = x[DIX(i - 1, j)];
			const float right = x[DIX(i + 1, j)];
			const float above = x[DIX(i, j + 1)];
			const float below = x[DIX(i, j - 1)];
			const float sum = left + right + above + below;
			const float prod = sum * a;
			const float addOrig = prod + x0[DIX(i, j)];
			const float result = addOrig / c;
			x[DIX(i, j)] = result;
		} END_JOBS
	}

	__global__
	void gpu_advect(float *d_d, float *d_d0, float *d_u, float *d_v, const float dt){
		uint32_t i = blockIdx.x*(JOBSPERTHREAD*blockDim.x)+threadIdx.x*JOBSPERTHREAD;
		uint32_t const j = blockIdx.y*blockDim.y+threadIdx.y;

		uint32_t i0, j0, i1, j1;
		float x, y, s0, t0, s1, t1, dt0;

		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			dt0 = dt * d_N;
			x = i - dt0 * d_u[DIX(i, j)];
			y = j - dt0 * d_v[DIX(i, j)];
			if (x < 0.5f)
				x = 0.5f;
			if (x > d_N + 0.5f)
				x = d_N + 0.5f;
			i0 = (int)x;
			i1 = i0 + 1;
			if (y < 0.5f)
				y = 0.5f;
			if (y > d_N + 0.5f)
				y = d_N + 0.5f;
			j0 = (int)y;
			j1 = j0 + 1;
			s1 = x - i0;
			s0 = 1 - s1;
			t1 = y - j0;
			t0 = 1 - t1;
			d_d[DIX(i, j)] = s0 * (t0 * d_d0[DIX(i0, j0)] + t1 * d_d0[DIX(i0, j1)]) +
				s1 * (t0 * d_d0[DIX(i1, j0)] + t1 * d_d0[DIX(i1, j1)]);
		} END_JOBS
	}

	__global__
	void gpu_set_bnd(uint32_t b, float *x)
	{
		uint32_t const n = blockIdx.x*blockDim.x+threadIdx.x;
				
		const uint32_t ub = d_N+d_pad;
		const uint32_t lb = d_pad-1;
		if(n < ub && n > lb){
			x[DIX(lb, n)] = (b == 1) ? -x[DIX(lb+1, n)] : x[DIX(lb+1, n)];	
			x[DIX(ub, n)] = (b == 1) ? -x[DIX(ub-1, n)] : x[DIX(ub-1, n)];
			x[DIX(n, lb)] = (b == 2) ? -x[DIX(n, lb+1)] : x[DIX(n, lb+1)];
			x[DIX(n, ub)] = (b == 2) ? -x[DIX(n, ub-1)] : x[DIX(n, ub-1)];

		}

		//x[DIX(d_pad-1, d_pad-1)] = 0.5f * (x[DIX(d_pad, d_pad-1)] + x[DIX(d_pad-1, d_pad)]);
		//x[DIX(d_pad-1, d_N + d_pad)] = 0.5f * (x[DIX(d_pad, d_N+d_pad)] + x[DIX(d_pad-1, d_N+d_pad-1)]);
		//x[DIX(N+d_pad, d_pad-1)] = 0.5f * (x[DIX(N+d_pad-1,d_pad-1)] + x[DIX(N+d_pad, d_pad)]);
		//x[DIX(N+d_pad, d_N+d_pad)] = 0.5f * (x[DIX(N+d_pad-1, d_N+d_pad)] + x[DIX(N+d_pad, d_N+d_pad-1)]);
	}

	__global__
	void gpu_add_source(float *x, float *s, const float dt){
		uint32_t i = blockIdx.x*(JOBSPERTHREAD*blockDim.x)+threadIdx.x*JOBSPERTHREAD;
		uint32_t const j = blockIdx.y*blockDim.y+threadIdx.y;

		FOR_JOBS_IN_BOUNDS { 
			DEVICE_CHECK_OOB
			x[DIX(i,j)] += dt * s[DIX(i,j)];
		} END_JOBS
	}

	__host__
	void add_source(float *x, float *s, float dt)
	{
		uint32_t i, size = array_size;
		for (i = 0; i < size; i++)
			x[i] += dt * s[i];
	}

	void lin_solve(uint32_t b, float *x, float *x0, float a, float c)
	{
		auto const size = sizeof(float)*array_size;
		float *dev_x, *dev_x0;
		gpuErrchk( hipMalloc((void**)&dev_x, size));
		gpuErrchk( hipMalloc((void**)&dev_x0, size));
		gpuErrchk( hipMemcpy(dev_x, x, size, hipMemcpyHostToDevice));
		gpuErrchk( hipMemcpy(dev_x0, x0, size, hipMemcpyHostToDevice));

		uint32_t k;
		for (k = 0; k < 20; k++)
		{
			CUDA::gpu_lin_solve<<<gridSize, threadCount>>>(dev_x, dev_x0, a, c);
			CUDA::gpu_set_bnd<<<bx, BSIZE>>>(b, dev_x);
		}

		gpuErrchk(hipMemcpy(x, dev_x, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(x0, dev_x0, size, hipMemcpyDeviceToHost));

		gpuErrchk(hipFree(dev_x));
		gpuErrchk(hipFree(dev_x0));
	}

	void lin_solve_2(uint32_t b, float *d_x, float *d_x0, float a, float c)
	{
		uint32_t k;
		for (k = 0; k < 20; k++)
		{
			CUDA::gpu_lin_solve<<<gridSize, threadCount>>>(d_x, d_x0, a, c);
			CUDA::gpu_set_bnd<<<bx, BSIZE>>>(b, d_x);
		}
	}

	void diffuse(uint32_t b, float *x, float *x0, float diff, float dt)
	{
		float a = dt * diff * N * N;
		CUDA::lin_solve(b, x, x0, a, 1 + 4 * a);
	}

	void diffuse_2(uint32_t b, float *d_x, float *d_x0, float diff, float dt)
	{
		float a = dt * diff * N * N;
		CUDA::lin_solve_2(b, d_x, d_x0, a, 1 + 4 * a);
	}

	void advect(uint32_t b, float *d, float *d0, float *u, float *v, float dt)
	{
		auto const size = sizeof(float)*array_size;

		float *d_d, *d_d0, *d_u, *d_v;			 
		gpuErrchk(hipMalloc((void**)&d_d, size	));
		gpuErrchk(hipMalloc((void**)&d_d0, size));
		gpuErrchk(hipMalloc((void**)&d_u, size	));
		gpuErrchk(hipMalloc((void**)&d_v, size	));

		gpuErrchk(hipMemcpy(d_d, d, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_d0, d0, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice	));

		CUDA::gpu_advect<<<gridSize, threadCount>>>(d_d, d_d0, d_u, d_v, dt);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(b, d_d);

		gpuErrchk(hipMemcpy(d, d_d, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(d_d));
		gpuErrchk(hipMemcpy(d0, d_d0, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(d_d0));
		gpuErrchk(hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(d_u));
		gpuErrchk(hipMemcpy(v, d_v, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(d_v));

	}

	void project(float *u, float *v, float *p, float *div){

		auto const size = sizeof(float)*array_size;

		float *d_u, *d_v, *d_p, *d_div;

		gpuErrchk(hipMalloc((void**)&d_v, size		));
		gpuErrchk(hipMalloc((void**)&d_p, size		));
		gpuErrchk(hipMalloc((void**)&d_u, size		));
		gpuErrchk(hipMalloc((void**)&d_div, size	));

		gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice		));
		gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice		));
		gpuErrchk(hipMemcpy(d_p, p, size, hipMemcpyHostToDevice		));
		gpuErrchk(hipMemcpy(d_div, div, size, hipMemcpyHostToDevice	));

		const float mul = -0.5f/(float)N;
		CUDA::gpu_project_1<<<gridSize, threadCount>>>(mul, d_u, d_v, d_p, d_div);

		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_div);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_p);

		uint32_t k;
		for (k = 0; k < 20; k++)
		{
			CUDA::gpu_lin_solve<<<gridSize, threadCount>>>(d_p, d_div, 1, 4);
			CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_p);
		}

		CUDA::gpu_project_2<<<gridSize, threadCount>>>(d_p, d_u, d_v);

		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(1, d_u);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(2, d_v);
																		 
		gpuErrchk(hipMemcpy(p, d_p, size, hipMemcpyDeviceToHost		));
		gpuErrchk(hipMemcpy(div, d_div, size, hipMemcpyDeviceToHost	));
		gpuErrchk(hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost		));
		gpuErrchk(hipMemcpy(v, d_v, size, hipMemcpyDeviceToHost		));

		gpuErrchk(hipFree(d_p	));
		gpuErrchk(hipFree(d_div));
		gpuErrchk(hipFree(d_u	));
		gpuErrchk(hipFree(d_v	));
	}

	void project_2(float *d_u, float *d_v, float *d_p, float *d_div)
	{
		const float mul = -0.5f/(float)N;
		CUDA::gpu_project_1<<<gridSize, threadCount>>>(mul, d_u, d_v, d_p, d_div);

		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_div);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_p);

		uint32_t k;
		for (k = 0; k < 20; k++)
		{
			CUDA::gpu_lin_solve<<<gridSize, threadCount>>>(d_p, d_div, 1, 4);
			CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_p);
		}

		CUDA::gpu_project_2<<<gridSize, threadCount>>>(d_p, d_u, d_v);

		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(1, d_u);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(2, d_v);
	}

	void dens_step(float *x, float *x0, float *u, float *v, float diff, float dt)
	{
	
		/*
			Prepare device memory
		*/

		auto const size = sizeof(float)*array_size;
		float *d_x, *d_x0, *d_u, *d_v;	
		gpuErrchk(hipMalloc((void**)&d_x, size));
		gpuErrchk(hipMalloc((void**)&d_x0, size));
		gpuErrchk(hipMalloc((void**)&d_u, size	));
		gpuErrchk(hipMalloc((void**)&d_v, size	));
		gpuErrchk(hipMemcpy(d_x, x, size, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_x0, x0, size, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice	));

		/*
			Compute solution
		*/
		
		CUDA::gpu_add_source<<<gridSize, threadCount>>>(d_x, d_x0, dt);
		SWAP(d_x0, d_x);
		CUDA::diffuse_2(0, d_x, d_x0, diff, dt);
		SWAP(d_x0, d_x);
		CUDA::gpu_advect<<<gridSize, threadCount>>>(d_x, d_x0, d_u, d_v, dt);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(0, d_u);

		/*
			Return result to CPU memory
		*/

		gpuErrchk(hipMemcpy(x, d_x, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(x0, d_x0, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(v, d_v, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(d_x));
		gpuErrchk(hipFree(d_x0));
		gpuErrchk(hipFree(d_u));
		gpuErrchk(hipFree(d_v));


	}
	
	void vel_step(float *u, float *v, float *u0, float *v0, float visc, float dt)
	{

		/*
			Prepare device memory
		*/
		auto const size = sizeof(float)*array_size;
		float *d_u, *d_v, *d_u0, *d_v0; 	
		float *d_v0_temp, *d_u0_temp; 	
		gpuErrchk(hipMalloc((void**)&d_u, size	));
		gpuErrchk(hipMalloc((void**)&d_v, size	));
		gpuErrchk(hipMalloc((void**)&d_u0, size));
		gpuErrchk(hipMalloc((void**)&d_v0, size));

		gpuErrchk(hipMalloc((void**)&d_v0_temp, size));
		gpuErrchk(hipMalloc((void**)&d_u0_temp, size));

		gpuErrchk(hipMemcpy(d_u, u, size, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_v, v, size, hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_u0, u0, size, hipMemcpyHostToDevice	));
		gpuErrchk(hipMemcpy(d_v0, v0, size, hipMemcpyHostToDevice	));


		/*
			Compute solution
		*/
		
		CUDA::gpu_add_source<<<gridSize, threadCount>>>(d_u, d_u0, dt);
		SWAP(d_u0, d_u);
		CUDA::diffuse_2(1, d_u, d_u0, visc, dt);
		
		CUDA::gpu_add_source<<<gridSize, threadCount>>>(d_v, d_v0, dt);
		SWAP(d_v0, d_v);
		CUDA::diffuse_2(2, d_v, d_v0, visc, dt);
		
		CUDA::project_2(d_u, d_v, d_u0, d_v0);
		
		SWAP(d_u0, d_u);
		SWAP(d_v0, d_v);

		gpuErrchk(hipMemcpy(d_u0_temp, d_u0, size, hipMemcpyDeviceToDevice ));
		gpuErrchk(hipMemcpy(d_v0_temp, d_v0, size, hipMemcpyDeviceToDevice ));
		
		CUDA::gpu_advect<<<gridSize, threadCount>>>(d_u, d_u0_temp, d_u0, d_v0, dt);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(1, d_u);

		CUDA::gpu_advect<<<gridSize, threadCount>>>(d_v, d_v0_temp, d_u0, d_v0, dt);
		CUDA::gpu_set_bnd<<<bx, BSIZE>>>(2, d_v);
		
		CUDA::project_2(d_u, d_v, d_u0, d_v0);

		/*
			Return result to CPU memory
		*/

		gpuErrchk(hipMemcpy(u, d_u, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(v, d_v, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(u0, d_u0, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(v0, d_v0, size, hipMemcpyDeviceToHost));
		gpuErrchk(hipFree(d_u));
		gpuErrchk(hipFree(d_v));
		gpuErrchk(hipFree(d_u0));
		gpuErrchk(hipFree(d_v0));
		gpuErrchk(hipFree(d_u0_temp));
		gpuErrchk(hipFree(d_v0_temp));

		
	}
}

